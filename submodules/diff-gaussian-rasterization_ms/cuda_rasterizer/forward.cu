#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>

namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* dc, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* direct_color = ((glm::vec3*)dc) + idx;
	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * direct_color[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[0] + SH_C1 * z * sh[1] - SH_C1 * x * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[3] +
				SH_C2[1] * yz * sh[4] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[5] +
				SH_C2[3] * xz * sh[6] +
				SH_C2[4] * (xx - yy) * sh[7];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[8] +
					SH_C3[1] * xy * z * sh[9] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[10] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[11] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[12] +
					SH_C3[5] * z * (xx - yy) * sh[13] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[14];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* dc,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,
	float2* rects,
	float2* points_xy_image,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	const bool* culling,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	if (culling[idx])
		return;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	float opacity = opacities[idx];
	constexpr float alpha_threshold = 1.0f/255.0f;
	const float opacity_power_threshold = log(opacity / alpha_threshold);
	const float extent = min(3.33, sqrt(2.0f * opacity_power_threshold));	

	float mid = 0.5f * (cov.x + cov.z);
	float lambda = mid + sqrt(max(0.01f, mid * mid - det));
	float my_radius = extent * sqrt(lambda);
	if (my_radius <= 0.0f)
		return;	
	
	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	const float extent_x = min(extent * sqrt(cov.x), my_radius);
	const float extent_y = min(extent * sqrt(cov.z), my_radius);
	const float2 rect_dims = make_float2(extent_x, extent_y);

	uint2 rect_min, rect_max;
	getRect(point_image, rect_dims, rect_min, rect_max, grid);	
	const int tile_count_rect = (rect_max.x - rect_min.x) * (rect_max.y - rect_min.y);
	if (tile_count_rect == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, dc, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	radii[idx] = (int) ceil(my_radius);
	rects[idx] = rect_dims;
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacity };
	tiles_touched[idx] = tile_count_rect;
}



// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	const uint32_t* __restrict__ per_tile_bucket_offset, uint32_t* __restrict__ bucket_to_tile,
	float* __restrict__ sampled_T, float* __restrict__ sampled_ar,
	int W, int H,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,

	const bool flag_max_count,
	float* __restrict__ accum_max_count,

	const float4* __restrict__ conic_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	uint32_t* __restrict__ max_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color
	)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint32_t tile_id = block.group_index().y * horizontal_blocks + block.group_index().x;
	uint2 range = ranges[tile_id];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// what is the number of buckets before me? what is my offset?
	uint32_t bbm = tile_id == 0 ? 0 : per_tile_bucket_offset[tile_id - 1];
	// let's first quickly also write the bucket-to-tile mapping
	int num_buckets = (toDo + 31) / 32;
	for (int i = 0; i < (num_buckets + BLOCK_SIZE - 1) / BLOCK_SIZE; ++i) {
		int bucket_idx = i * BLOCK_SIZE + block.thread_rank();
		if (bucket_idx < num_buckets) {
			bucket_to_tile[bbm + bucket_idx] = tile_id;
		}
	}
	


	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };

	float weight_max=0;

	int idx_max=0;
	int flag_update=0;

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// add incoming T value for every 32nd gaussian
			if (j % 32 == 0) {
				sampled_T[(bbm * BLOCK_SIZE) + block.thread_rank()] = T;
				for (int ch = 0; ch < CHANNELS; ++ch) {
					sampled_ar[(bbm * BLOCK_SIZE * CHANNELS) + ch * BLOCK_SIZE + block.thread_rank()] = C[ch];
				}
				++bbm;
			}			

			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;


			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;

			if(weight_max<alpha * T)
			{
				weight_max=alpha * T;
				idx_max = collected_id[j];
				flag_update = 1;
			}

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	if(flag_update==1 && flag_max_count)
	{
		atomicAdd(&(accum_max_count[idx_max]), 1);
	}


	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];	
	}

	// max reduce the last contributor
    typedef hipcub::BlockReduce<uint32_t, BLOCK_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_Y> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    last_contributor = BlockReduce(temp_storage).Reduce(last_contributor, hipcub::Max());
	if (block.thread_rank() == 0) {
		max_contrib[tile_id] = last_contributor;
	}	
}


template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
render_simpCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,

	float* __restrict__ accum_weights_p,
	int* __restrict__ accum_weights_count,
	float* __restrict__ accum_max_count,

	const float4* __restrict__ conic_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color
	)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint32_t tile_id = block.group_index().y * horizontal_blocks + block.group_index().x;
	uint2 range = ranges[tile_id];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };

	float weight_max=0;

	int idx_max=0;
	int flag_update=0;

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{	

			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;

			if(weight_max<alpha * T)
			{
				weight_max=alpha * T;
				idx_max = collected_id[j];
				flag_update = 1;
			}

			atomicAdd(&(accum_weights_p[collected_id[j]]), alpha * T);
			atomicAdd(&(accum_weights_count[collected_id[j]]), 1);
			
			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	if(flag_update==1)
	{
		atomicAdd(&(accum_max_count[idx_max]), 1);
	}


	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];	
	}
}






template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
render_depthCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,

	const float4* __restrict__ conic_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,

	float* __restrict__ out_color,
	float* __restrict__ out_pts,
	float* __restrict__ out_depth,
	float* accum_alpha,
	int* __restrict__ gidx,
	float* __restrict__ discriminants,

	const float* __restrict__ means3D,
	const glm::vec3* __restrict__ scales,
	const glm::vec4* __restrict__ rotations,

	const float* __restrict__ viewmatrix,
	const float* __restrict__ projmatrix,
	const glm::vec3* __restrict__ cam_pos
	)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };

	float weight_max=0;
	float depth_max=0;
	float discriminant_max=0;

	int idx_max=0;
	int flag_update=0;

    glm::mat4 matrix = glm::make_mat4x4(projmatrix);
    glm::mat4 matrix_temp = glm::inverse(matrix);
	float *projmatrix_inv= glm::value_ptr(matrix_temp);

	glm::vec3 ray_origin = *cam_pos;
	glm::vec3 point_rec = {0,0,0};





	float3 p_proj_r = { Pix2ndc(pixf.x, W), Pix2ndc(pixf.y, H), 1};

	//inverse process of 'Transform point by projecting'
	float p_hom_x_r = p_proj_r.x*(1.0000001);
	float p_hom_y_r = p_proj_r.y*(1.0000001);
	// self.zfar = 100.0, self.znear = 0.01
	float p_hom_z_r = (100-100*0.01)/(100-0.01);
	float p_hom_w_r = 1;


	float3 p_hom_r={p_hom_x_r, p_hom_y_r, p_hom_z_r};
	float4 p_orig_r=transformPoint4x4(p_hom_r, projmatrix_inv);

	glm::vec3 ray_direction={
		p_orig_r.x-ray_origin.x,
		p_orig_r.y-ray_origin.y,
		p_orig_r.z-ray_origin.z,
	};
	glm::vec3 normalized_ray_direction = glm::normalize(ray_direction);




	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
		}
		block.sync();

	
		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{

	
			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;

			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}	

			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;
				
			// compute Gaussian depth
			// Normalize quaternion to get valid rotation
			glm::vec4 q = rotations[collected_id[j]];// / glm::length(rot);
			float rot_r = q.x;
			float rot_x = q.y;
			float rot_y = q.z;
			float rot_z = q.w;


			// Compute rotation matrix from quaternion
			glm::mat3 R = glm::mat3(
				1.f - 2.f * (rot_y * rot_y + rot_z * rot_z), 2.f * (rot_x * rot_y - rot_r * rot_z), 2.f * (rot_x * rot_z + rot_r * rot_y),
				2.f * (rot_x * rot_y + rot_r * rot_z), 1.f - 2.f * (rot_x * rot_x + rot_z * rot_z), 2.f * (rot_y * rot_z - rot_r * rot_x),
				2.f * (rot_x * rot_z - rot_r * rot_y), 2.f * (rot_y * rot_z + rot_r * rot_x), 1.f - 2.f * (rot_x * rot_x + rot_y * rot_y)
			);


			glm::vec3 temp={
				ray_origin.x-means3D[3*collected_id[j]+0],
				ray_origin.y-means3D[3*collected_id[j]+1],
				ray_origin.z-means3D[3*collected_id[j]+2],
			};
			glm::vec3 rotated_ray_origin = R * temp;
			glm::vec3 rotated_ray_direction = R * normalized_ray_direction;


			glm::vec3 a_t= rotated_ray_direction/(scales[collected_id[j]]*3.0f)*rotated_ray_direction/(scales[collected_id[j]]*3.0f);
			float a = a_t.x + a_t.y + a_t.z;

			glm::vec3 b_t= rotated_ray_direction/(scales[collected_id[j]]*3.0f)*rotated_ray_origin/(scales[collected_id[j]]*3.0f);
			float b = 2*(b_t.x + b_t.y + b_t.z);

			glm::vec3 c_t= rotated_ray_origin/(scales[collected_id[j]]*3.0f)*rotated_ray_origin/(scales[collected_id[j]]*3.0f);
			float c = c_t.x + c_t.y + c_t.z-1;


			float discriminant=b*b-4*a*c;	


			float depth = (-b/2/a)/glm::length(ray_direction);
			

			if(depth<0)
				continue;



			if(weight_max<alpha * T)
			{
				weight_max=alpha * T;
				depth_max=depth;
				discriminant_max=discriminant;
				idx_max=collected_id[j];

				point_rec = ray_origin+(-b/2/a)*normalized_ray_direction;			
			}

		
			
			T = test_T;
			last_contributor = contributor;
		}		
			

	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		for (int ch = 0; ch < 3; ch++)
			out_pts[ch * H * W + pix_id] = point_rec[ch];

		out_depth[pix_id] = depth_max;
		accum_alpha[pix_id] = T;
		discriminants[pix_id] = discriminant_max;
		gidx[pix_id]=idx_max;
	}
}










void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	const uint32_t* per_tile_bucket_offset, uint32_t* bucket_to_tile,
	float* sampled_T, float* sampled_ar,	
	int W, int H,
	const float2* means2D,
	const float* colors,

	const bool flag_max_count,
	float* accum_max_count,
	
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	uint32_t* max_contrib,
	const float* bg_color,
	float* out_color
	)
{
	renderCUDA<NUM_CHAFFELS> << <grid, block >> > (
		ranges,
		point_list,
		per_tile_bucket_offset, bucket_to_tile,
		sampled_T, sampled_ar,		
		W, H,
		means2D,
		colors,
		flag_max_count,
		accum_max_count,
		conic_opacity,
		final_T,
		n_contrib,
		max_contrib,
		bg_color,
		out_color
		);
}




void FORWARD::render_simp(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float2* means2D,
	const float* colors,
	float* accum_weights_p,
	int* accum_weights_count,
	float* accum_max_count,
	
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color
	)
{
	render_simpCUDA<NUM_CHAFFELS> << <grid, block >> > (
		ranges,
		point_list,	
		W, H,
		means2D,
		colors,
		accum_weights_p,	
		accum_weights_count,
		accum_max_count,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color
		);
}


void FORWARD::render_depth(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float2* means2D,
	const float* colors,
	
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,

	float* out_color,
	float* out_pts,
	float* out_depth,
	float* accum_alpha,
	int* gidx,
	float* discriminants,

	const float* means3D,
	const glm::vec3* scales,
	const glm::vec4* rotations,

	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos
	)
{
	render_depthCUDA<NUM_CHAFFELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		means2D,
		colors,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color,
		out_pts,
		
		out_depth,
		accum_alpha,
		gidx,
		discriminants,

		means3D,
		scales,
		rotations,

		viewmatrix, 
		projmatrix,
		cam_pos	
		);
}




void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* dc,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float2* rects,
	float2* means2D,
	float* depths,

	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	const bool* culling,
	bool prefiltered)
{
	preprocessCUDA<NUM_CHAFFELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		dc,
		shs,
		clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		rects,
		means2D,
		depths,

		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		culling,
		prefiltered
		);
}



